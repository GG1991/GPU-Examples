#include "hip/hip_runtime.h"

#include "ell.hpp"
#include "vars.hpp"

#include "hip/hip_runtime_api.h"

#define cudaCheckError() { \
         hipError_t e=hipGetLastError(); \
	 if(e!=hipSuccess) { \
		    printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
		    exit(0); \
		  } \
}

#include <iostream>
#include <chrono>
using namespace std;
using namespace std::chrono;

#ifdef CPU
void get_ctan(const double *eps, double *ctan, const double *history_params)
{
	const double lambda = 1.0e1;
	const double mu = 1.3e5;

	memset(ctan, 0, 6 * 6 * sizeof(double));

	for (int i = 0; i < 3; ++i)
		for (int j = 0; j < 3; ++j)
			ctan[i * 6 + j] += lambda;

	for (int i = 0; i < 3; ++i)
		ctan[i * 6 + i] += 2 * mu;

	for (int i = 3; i < 6; ++i)
		ctan[i * 6 + i] = mu;
}


void get_elem_nodes(int n[NPE], const int nx, const int ny, const int nz, int ex, int ey, int ez)
{
	const int nxny = ny * nx;
	const int n0 = ez * nxny + ey * nx + ex;
	n[0] = n0;
	n[1] = n0 + 1;
	n[2] = n0 + nx + 1;
	n[3] = n0 + nx;

	if (DIM == 3) {
		n[4] = n[0] + nxny;
		n[5] = n[1] + nxny;
		n[6] = n[2] + nxny;
		n[7] = n[3] + nxny;
	}
}


void get_elem_displ(const double *u, double elem_disp[NPE * DIM], const int nx, const int ny, const int nz, int ex, int ey, int ez)
{
	int n[NPE];
	get_elem_nodes(n, nx, ny, nz, ex, ey, ez);

	for (int i = 0 ; i < NPE; ++i) {
		for (int d = 0; d < DIM; ++d) {
			elem_disp[i * DIM + d] = u[n[i] * DIM + d];
		}
	}
}


void get_strain(const double *u, int gp, double *strain_gp,
	       	CUDA_vars *CUDA_vars_h,
		int ex, int ey, int ez)
{
	double elem_disp[NPE * DIM];
	const int nx = CUDA_vars_h->nx;
	const int ny = CUDA_vars_h->ny;
	const int nz = CUDA_vars_h->nz;
	get_elem_displ(u, elem_disp, nx, ny, nz, ex, ey, ez);

	for (int i = 0; i < NVOI; ++i) {
		strain_gp[i] = 0;
	}

	for (int v = 0; v < NVOI; ++v) {
		for (int i = 0; i < NPE * DIM; ++i){
			strain_gp[v] += CUDA_vars_h->bmat_cache[gp][v][i] * elem_disp[i];
		}
	}
}


void get_elem_mat(const double *u, double Ae[NPE * DIM * NPE * DIM],
	       	  CUDA_vars *CUDA_vars_h,
		  int ex, int ey, int ez)
{
	const double wg = 0.25;
	double ctan[NVOI][NVOI];
	constexpr int npedim = NPE * DIM;
	constexpr int npedim2 = npedim * npedim;

	double TAe[npedim2] = { 0.0 };

	for (int gp = 0; gp < NPE; ++gp) {

		double eps[6];
		get_strain(u, gp, eps, CUDA_vars_h, ex, ey, ez);

		get_ctan(eps, (double *)ctan, nullptr);

		double cxb[NVOI][npedim];

		for (int i = 0; i < NVOI; ++i) {
			for (int j = 0; j < npedim; ++j) {
				double tmp = 0.0;
				for (int k = 0; k < NVOI; ++k)
					tmp += ctan[i][k] * CUDA_vars_h->bmat_cache[gp][k][j];
				cxb[i][j] = tmp * wg;
			}
		}

		for (int m = 0; m < NVOI; ++m) {
			for (int i = 0; i < npedim; ++i) {
				const int inpedim = i * npedim;
				const double bmatmi = CUDA_vars_h->bmat_cache[gp][m][i];
				for (int j = 0; j < npedim; ++j)
					TAe[inpedim + j] += bmatmi * cxb[m][j];
			}
		}
	}
	memcpy(Ae, TAe, npedim2 * sizeof(double));
}

void get_elem_mats_cpu(double *Ae_arr, const double *ctan_arr,
		       CUDA_vars *CUDA_vars_h)
{
	const double wg = 0.25;
	const int npedim = NPE * DIM;
	const int npedim2 = npedim * npedim;
	const int nex = CUDA_vars_h->nex;
	const int ney = CUDA_vars_h->ney;
	const int nez = CUDA_vars_h->nez;

	for (int ex = 0; ex < nex; ++ex) {
		for (int ey = 0; ey < ney; ++ey) {
			for (int ez = 0; ez < nez; ++ez) {
	double TAe[npedim2] = { 0.0 };
	for (int gp = 0; gp < NPE; ++gp) {

const double *ctan = &ctan_arr[glo_elem(ex,ey,ez) * NPE * NVOI2
			+ gp * NVOI2];
		double cxb[NVOI][npedim];

		for (int i = 0; i < NVOI; ++i) {
			for (int j = 0; j < npedim; ++j) {
				double tmp = 0.0;
				for (int k = 0; k < NVOI; ++k)
					tmp += ctan[i * NVOI + k] 
						* CUDA_vars_h->bmat_cache[gp][k][j];
				cxb[i][j] = tmp * wg;
			}
		}

		for (int m = 0; m < NVOI; ++m) {
			for (int i = 0; i < npedim; ++i) {
				const int inpedim = i * npedim;
				const double bmatmi = CUDA_vars_h->bmat_cache[gp][m][i];
				for (int j = 0; j < npedim; ++j)
					TAe[inpedim + j] += bmatmi * cxb[m][j];
			}
		}
	}
	double *Ae = &Ae_arr[glo_elem(ex,ey,ez) * NPEDIM2];
	memcpy(Ae, TAe, NPEDIM2 * sizeof(double));
			}
		}
	}
}


void assembly_mat(ell_matrix *A, const double *u, CUDA_vars *CUDA_vars_h)
{
	ell_set_zero_mat(A);

	const int nex = CUDA_vars_h->nex;
	const int ney = CUDA_vars_h->ney;
	const int nez = CUDA_vars_h->nez;

	double Ae[NPE * DIM * NPE * DIM];
	for (int ex = 0; ex < nex; ++ex) {
		for (int ey = 0; ey < ney; ++ey) {
			for (int ez = 0; ez < nez; ++ez) {
				get_elem_mat(u, Ae, CUDA_vars_h, ex, ey, ez);
				ell_add_3D(A, ex, ey, ez, Ae);
			}
		}
	}
	//ell_set_bc_3D(A);
}
#endif


#ifdef GPU
__device__
void get_ctan_d(const double *eps, double *ctan, const double *history_params)
{
	const double lambda = 1.0e1;
	const double mu = 1.3e5;

	memset(ctan, 0, 6 * 6 * sizeof(double));

	for (int i = 0; i < 3; ++i)
		for (int j = 0; j < 3; ++j)
			ctan[i * 6 + j] += lambda;

	for (int i = 0; i < 3; ++i)
		ctan[i * 6 + i] += 2 * mu;

	for (int i = 3; i < 6; ++i)
		ctan[i * 6 + i] = mu;
}


__device__
void get_elem_nodes_d(int n[NPE], const int nx, const int ny, const int nz, int ex, int ey, int ez)
{
	const int nxny = ny * nx;
	const int n0 = ez * nxny + ey * nx + ex;
	n[0] = n0;
	n[1] = n0 + 1;
	n[2] = n0 + nx + 1;
	n[3] = n0 + nx;

	if (DIM == 3) {
		n[4] = n[0] + nxny;
		n[5] = n[1] + nxny;
		n[6] = n[2] + nxny;
		n[7] = n[3] + nxny;
	}
}


__device__
void get_elem_displ_d(const double *u, double elem_disp[NPE * DIM], const int nx, const int ny, const int nz, int ex, int ey, int ez)
{
	int n[NPE];
	get_elem_nodes_d(n, nx, ny, nz, ex, ey, ez);

	for (int i = 0 ; i < NPE; ++i) {
		for (int d = 0; d < DIM; ++d) {
			elem_disp[i * DIM + d] = u[n[i] * DIM + d];
		}
	}
}


__device__
void get_strain_d(const double *u, int gp, double *strain_gp,
	       	CUDA_vars *CUDA_vars_d,
		int ex, int ey, int ez)
{
	const int nx = CUDA_vars_d->nx;
	const int ny = CUDA_vars_d->ny;
	const int nz = CUDA_vars_d->nz;
	double elem_disp[NPE * DIM];
	get_elem_displ_d(u, elem_disp, nx, ny, nz, ex, ey, ez);

	for (int i = 0; i < NVOI; ++i) {
		strain_gp[i] = 0;
	}

	for (int v = 0; v < NVOI; ++v) {
		for (int i = 0; i < NPE * DIM; ++i){
			strain_gp[v] += CUDA_vars_d->bmat_cache[gp][v][i] * elem_disp[i];
		}
	}
}

__device__
void ell_add_3D_gpu(ell_matrix *m, double *vals_d, int ex, int ey, int ez, const double Ae[NPEDIM2])
{
	// assembly Ae in 3D structured grid representation
	// nFields : number of scalar components on each node

	const int nx = m->n[0];
	const int ny = m->n[1];
	const int nfield = m->nfield;
	const int npe = 8;
	const int nnz = m->nnz;
	const int cols_row[8][8] = {
		{ 13, 14, 17, 16, 22, 23, 26, 25 },
		{ 12, 13, 16, 15, 21, 22, 25, 24 },
		{ 9,  10, 13, 12, 18, 19, 22, 21 },
		{ 10, 11, 14, 13, 19, 20, 23, 22 },
		{ 4,  5,  8,  7,  13, 14, 17, 16 },
		{ 3,  4,  7,  6,  12, 13, 16, 15 },
		{ 0,  1,  4,  3,  9,  10, 13, 12 },
		{ 1,  2,  5,  4,  10, 11, 14, 13 } };

	const int nxny = nx * ny;
	const int n0 = ez * nxny + ey * nx + ex;
	const int n1 = n0 + 1;
	const int n2 = n0 + nx + 1;
	const int n3 = n0 + nx;

	const int ix_glo[8] = {	n0, n1, n2, n3,
		n0 + nxny,
		n1 + nxny,
		n2 + nxny,
		n3 + nxny };

	const int nnz_nfield = nfield * nnz;
	const int npe_nfield = npe * nfield;
	const int npe_nfield2 = npe * nfield * nfield;

	for (int fi = 0; fi < nfield; ++fi)
		for (int fj = 0; fj < nfield; ++fj)
			for (int i = 0; i < npe; ++i)
				for (int j = 0; j < npe; ++j){
					vals_d[ix_glo[i] * nnz_nfield + cols_row[i][j] * nfield + fi * nnz + fj] +=
						Ae[i * npe_nfield2 + fi * npe_nfield + j * nfield + fj];
					__syncthreads();
				}

}


__global__
void get_elem_mats_gpu(double *Ae_arr, const double *u,
		       	 CUDA_vars *CUDA_vars_d)
{
	const double wg = 0.25;
	const int npedim = NPE * DIM;
	const int npedim2 = npedim * npedim;
	const int nex = CUDA_vars_d->nex;
	const int ney = CUDA_vars_d->ney;
	const int nez = CUDA_vars_d->nez;

	int ex_t = threadIdx.x + blockDim.x * blockIdx.x;
	int ey_t = threadIdx.y + blockDim.y * blockIdx.y;
	int ez_t = threadIdx.z + blockDim.z * blockIdx.z;
	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;
	int stride_z = blockDim.z * gridDim.z;

	for (int ex = ex_t; ex < nex; ex += stride_x) {
		for (int ey = ey_t; ey < ney; ey += stride_y) {
			for (int ez = ez_t; ez < nez; ez += stride_z) {
	double TAe[npedim2] = { 0.0 };
	for (int gp = 0; gp < NPE; ++gp) {

		double eps[6];
		double ctan[NVOI2];
		get_strain_d(u, gp, eps, CUDA_vars_d, ex, ey, ez);
		get_ctan_d(eps, ctan, nullptr);
		double cxb[NVOI][npedim];

		for (int i = 0; i < NVOI; ++i) {
			for (int j = 0; j < npedim; ++j) {
				double tmp = 0.0;
				for (int k = 0; k < NVOI; ++k)
					tmp += ctan[i * NVOI + k] 
						* CUDA_vars_d->bmat_cache[gp][k][j];
				cxb[i][j] = tmp * wg;
			}
		}

		for (int m = 0; m < NVOI; ++m) {
			for (int i = 0; i < npedim; ++i) {
				const int inpedim = i * npedim;
				const double bmatmi = CUDA_vars_d->bmat_cache[gp][m][i];
				for (int j = 0; j < npedim; ++j)
					TAe[inpedim + j] += bmatmi * cxb[m][j];
			}
		}
	}
	double *Ae = &Ae_arr[glo_elem(ex,ey,ez) * NPEDIM2];
	memcpy(Ae, TAe, npedim2 * sizeof(double));
			}
		}
	}
}

__global__
void assembly_elem_mats_gpu(ell_matrix *A_d, double *vals_d, const double *u,
		       	 CUDA_vars *CUDA_vars_d)
{
	const double wg = 0.25;
	const int npedim = NPE * DIM;
	const int nex = CUDA_vars_d->nex;
	const int ney = CUDA_vars_d->ney;
	const int nez = CUDA_vars_d->nez;

	int ex_t = threadIdx.x + blockDim.x * blockIdx.x;
	int ey_t = threadIdx.y + blockDim.y * blockIdx.y;
	int ez_t = threadIdx.z + blockDim.z * blockIdx.z;
	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;
	int stride_z = blockDim.z * gridDim.z;

	for (int ex = ex_t; ex < nex; ex += stride_x) {
		for (int ey = ey_t; ey < ney; ey += stride_y) {
			for (int ez = ez_t; ez < nez; ez += stride_z) {
	double TAe[NPEDIM2] = { 0.0 };
	for (int gp = 0; gp < NPE; ++gp) {

		double eps[6];
		double ctan[NVOI2];
		get_strain_d(u, gp, eps, CUDA_vars_d, ex, ey, ez);
		get_ctan_d(eps, ctan, nullptr);
		double cxb[NVOI][npedim];

		for (int i = 0; i < NVOI; ++i) {
			for (int j = 0; j < npedim; ++j) {
				double tmp = 0.0;
				for (int k = 0; k < NVOI; ++k)
					tmp += ctan[i * NVOI + k] 
						* CUDA_vars_d->bmat_cache[gp][k][j];
				cxb[i][j] = tmp * wg;
			}
		}

		for (int m = 0; m < NVOI; ++m) {
			for (int i = 0; i < npedim; ++i) {
				const int inpedim = i * npedim;
				const double bmatmi = CUDA_vars_d->bmat_cache[gp][m][i];
				for (int j = 0; j < npedim; ++j)
					TAe[inpedim + j] += bmatmi * cxb[m][j];
			}
		}
	}
	//double *Ae = &Ae_arr[glo_elem(ex,ey,ez) * NPEDIM2];
	//memcpy(Ae, TAe, npedim2 * sizeof(double));
				ell_add_3D_gpu(A_d, vals_d, ex, ey, ez, TAe);
			}
		}
	}
}

void assembly_mat_gpu(ell_matrix *A, const double *u,
		      CUDA_vars *CUDA_vars_h)
{
	auto time_1 = high_resolution_clock::now();
	hipProfilerStart();
	ell_set_zero_mat(A);
	CUDA_vars *CUDA_vars_d;

	hipMalloc((void **)&CUDA_vars_d, sizeof(CUDA_vars));
	hipMemcpy(CUDA_vars_d, CUDA_vars_h, sizeof(CUDA_vars),
		   hipMemcpyHostToDevice);

	const int nex = CUDA_vars_h->nex;
	const int ney = CUDA_vars_h->ney;
	const int nez = CUDA_vars_h->nez;
	const int ne = nex * ney * nez;
	const int nx = CUDA_vars_h->nx;
	const int ny = CUDA_vars_h->ny;
	const int nz = CUDA_vars_h->nz;
	const int nn = nx * ny * nz;

	double *Ae_arr = new double[ne * NPEDIM2];

	double *u_d;
	double *Ae_arr_d;

	hipMalloc((void**)&Ae_arr_d, ne * NPEDIM2 * sizeof(double));
	hipMalloc((void**)&u_d, nn * DIM * sizeof(double));
	hipMemcpy(u_d, u,
		   nn * DIM * sizeof(double), 
		   hipMemcpyHostToDevice);

	dim3 grid(15, 15, 15);
	dim3 block(4, 4, 4);
	get_elem_mats_gpu<<<grid, block>>>(Ae_arr_d, u_d, 
					   CUDA_vars_h);
        cudaCheckError();
	hipMemcpy(Ae_arr, Ae_arr_d, 
		   ne * NPEDIM2 * sizeof(double),
		   hipMemcpyDeviceToHost);

	hipFree(Ae_arr_d);
	hipFree(u_d);
	hipFree(CUDA_vars_d);

	auto time_2 = high_resolution_clock::now();

	for (int ex = 0; ex < nex; ++ex) {
		for (int ey = 0; ey < ney; ++ey) {
			for (int ez = 0; ez < nez; ++ez) {
	double *Ae = &Ae_arr[glo_elem(ex,ey,ez) * NPEDIM2];
				ell_add_3D(A, ex, ey, ez, Ae);
			}
		}
	}

	delete [] Ae_arr;

	//ell_set_bc_3D(A);
	hipProfilerStop();
	auto time_3 = high_resolution_clock::now();

	auto duration = duration_cast<milliseconds>(time_2 - time_1);
	cout << "assembly 1 = " << duration.count() << " ms" << endl;
	duration = duration_cast<milliseconds>(time_3 - time_2);
	cout << "assembly 2 = " << duration.count() << " ms" << endl;
}

void assembly_mat_gpu_2(ell_matrix *A, const double *u,
		        CUDA_vars *CUDA_vars_h)
{
	auto time_1 = high_resolution_clock::now();
	hipProfilerStart();
	ell_set_zero_mat(A);
	CUDA_vars *CUDA_vars_d;

	hipMalloc((void **)&CUDA_vars_d, sizeof(CUDA_vars));
	hipMemcpy(CUDA_vars_d, CUDA_vars_h, sizeof(CUDA_vars),
		   hipMemcpyHostToDevice);

	const int nx = CUDA_vars_h->nx;
	const int ny = CUDA_vars_h->ny;
	const int nz = CUDA_vars_h->nz;
	const int nn = nx * ny * nz;

	double *u_d;
	double *vals_d;
	ell_matrix *A_d;

	hipMalloc((void**)&A_d, sizeof(ell_matrix));
	hipMalloc((void**)&vals_d, A->nnz * A->nrow * sizeof(double));
	hipMalloc((void**)&u_d, nn * DIM * sizeof(double));
	hipMemcpy(u_d, u,
		   nn * DIM * sizeof(double), 
		   hipMemcpyHostToDevice);
	hipMemcpy(A_d, A,
		   sizeof(ell_matrix), 
		   hipMemcpyHostToDevice);
	hipMemset(vals_d, 0, A->nnz * A->nrow * sizeof(double));

	dim3 grid(15, 15, 15);
	dim3 block(4, 4, 4);
	assembly_elem_mats_gpu<<<grid, block>>>(A_d, vals_d, u_d, 
					   CUDA_vars_h);
        cudaCheckError();
	hipMemcpy(A->vals, vals_d, 
		   A->nrow * A->nnz * sizeof(double),
		   hipMemcpyDeviceToHost);

	hipFree(A_d);
	hipFree(vals_d);
	hipFree(u_d);
	hipFree(CUDA_vars_d);

	auto time_2 = high_resolution_clock::now();

	//ell_set_bc_3D(A);
	hipProfilerStop();
	auto time_3 = high_resolution_clock::now();

	auto duration = duration_cast<milliseconds>(time_2 - time_1);
	cout << "assembly 1 = " << duration.count() << " ms" << endl;
	duration = duration_cast<milliseconds>(time_3 - time_2);
	cout << "assembly 2 = " << duration.count() << " ms" << endl;
}
#endif

#ifdef CPU
void assembly_mat_new_cpu(ell_matrix *A, const double *u, 
			  CUDA_vars *CUDA_vars_h)
{
	ell_set_zero_mat(A);

	const int nex = CUDA_vars_h->nex;
	const int ney = CUDA_vars_h->ney;
	const int nez = CUDA_vars_h->nez;
	const int ne = nex * ney * nez;

	double *ctan_arr = new double[ne * NPE * NVOI2];

	for (int ex = 0; ex < nex; ++ex) {
		for (int ey = 0; ey < ney; ++ey) {
			for (int ez = 0; ez < nez; ++ez) {
				for (int gp = 0; gp < NPE; ++gp) {
					double eps[6];
	double *ctan = &ctan_arr[glo_elem(ex,ey,ez) * NPE * NVOI2 
		+ gp * NVOI2];
					get_strain(u, gp, eps, CUDA_vars_h, ex, ey, ez);
					get_ctan(eps, ctan, nullptr);
				}
			}
		}
	}

	double *Ae_arr = new double[ne * NPEDIM2];

	get_elem_mats_cpu(Ae_arr, ctan_arr, CUDA_vars_h);

	for (int ex = 0; ex < nex; ++ex) {
		for (int ey = 0; ey < ney; ++ey) {
			for (int ez = 0; ez < nez; ++ez) {
	double *Ae = &Ae_arr[glo_elem(ex,ey,ez) * NPEDIM2];
				ell_add_3D(A, ex, ey, ez, Ae);
			}
		}
	}

	delete [] ctan_arr;
	delete [] Ae_arr;

	//ell_set_bc_3D(A);
}
#endif
